#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "HWAccelCUDA.cuh"


#if 0
}	// indent guard
#endif

void Legacy(int* in, int* out, int n)
{
	for (int i = 0; i < n; i++) {
		out[i] = in[i] * 2;
	}
}

__global__ void reduce0(double* g_idata, double* g_odata) {
	extern __shared__ double sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


__global__ void CalculateTerm(double* dAA, double* dBB, double* dSS, double* dCC, double* dRR, int n, double tj, double tit)
{
	int i = threadIdx.x;
	if (i < n) {
		double u = dAA[i] + (dBB[i] * tj);
		double su = sin(u);
		double cu = cos(u);
		dRR[i] = tit * (dSS[i] * su + dCC[i] * cu);
	}
}

double SUM(double* hIn, int n)
{
	double* dIn;
	double* dOut;
	double* hOut;
	int power = 0;
	int gridSize = n;
	while (true) {
		if (gridSize > 256) {
			power += 1;
			gridSize /= 256;
			continue;
		}
		break;
	}

	int blockSize = ((n / 256) + 1) * 256;
	gridSize = (n / 256) + 1;
	hipMalloc((void**)&dIn, blockSize * sizeof(double));
	hipMalloc((void**)&dOut, blockSize * sizeof(double));
	hipHostMalloc((void**)&hOut, blockSize * sizeof(double));
	hipMemcpy(dIn, hIn, n * sizeof(double), hipMemcpyHostToDevice);

	hipMemset(dIn, 0, blockSize * sizeof(double));

	
	while (true)
	{
		reduce0 <<<gridSize, 256>>> (dIn, dOut);
		hipMemset(dIn, 0, blockSize * sizeof(double));
		hipMemcpy(dIn, dOut, gridSize * sizeof(double), hipMemcpyDeviceToDevice);
		if (gridSize == 1) {
			break;
		}
		gridSize = gridSize / 256 + 1;
	}
	reduce0 <<<1, 256 >>> (dIn, dOut);
	hipDeviceSynchronize();
	hipMemcpy(hOut, dOut, gridSize * sizeof(double), hipMemcpyDeviceToHost);
	double result = hOut[0];
	hipFree(dIn);
	hipFree(dOut);
	hipFree(hOut);
	return result;
}

void CUDA(double* hAA, double* hBB,double* hSS,double* hCC, double* hRR, int n,double tj,double tit)
{
	int blocksize = (n / 256) + 1;
	int cudasize = blocksize * 256;

	double* dAA;
	double* dBB;
	double* dSS;
	double* dCC;
	double* dRR;

	hipMalloc((void**)&dAA, cudasize * sizeof(double));
	hipMalloc((void**)&dBB, cudasize * sizeof(double));
	hipMalloc((void**)&dSS, cudasize * sizeof(double));
	hipMalloc((void**)&dCC, cudasize * sizeof(double));
	hipMalloc((void**)&dRR, cudasize * sizeof(double));


	hipMemcpy(dAA, hAA, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dBB, hBB, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dSS, hSS, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dCC, hCC, n * sizeof(double), hipMemcpyHostToDevice);
	
	hipMemset(dRR, 0, cudasize * sizeof(double));

	CalculateTerm<<<blocksize, 256>>> (dAA,dBB,dSS,dCC,dRR,n,tj,tit);
	hipDeviceSynchronize();

	hipMemcpy(hRR, dRR, n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dAA);
	hipFree(dBB);
	hipFree(dSS);
	hipFree(dCC);
	hipFree(dRR);
}
